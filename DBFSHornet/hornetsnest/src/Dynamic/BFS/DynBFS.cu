#include "hip/hip_runtime.h"
#include "Dynamic/BFS/DynBFS.cuh"

// using BfsTopDown2Dynamic = BfsTopDown2<HornetDynamicGraph>;
// using BfsTopDown2Static  = BfsTopDown2<HornetStaticGraph>;

namespace hornets_nest 
{

    const dist_t INF = std::numeric_limits<dist_t>::max()-1;

//------------------------------------------------------------------------------
///////////////
// OPERATORS //
///////////////

struct ResetDistance
{
    dist_t* d_distances;

    OPERATOR(Vertex& vertex)
    {
        auto child = vertex.id();

        if(d_distances[child] != 0)
            d_distances[child] = INF;
    }

};

struct FatherUpdate
{
    dist_t* d_distances;
    dist_t* old_distances;

    OPERATOR(Vertex& vertex, Edge& edge)
    {
        auto child = vertex.id();
        auto father = edge.dst_id();

        // printf("BEFORE child %d childdeg %d father %d childd %d fatherd %d\n",child,vertex.degree(),father,old_distances[child],old_distances[father]);
        atomicMin(d_distances+child, old_distances[father]+1); //operation to find the father nearest to the source)
        // printf("AFTER child %d childdeg %d father %d childd %d fatherd %d\n",child,vertex.degree(),father,d_distances[child],old_distances[father]);

    }
};

struct LowestFather
{
    dist_t* d_distances;
    dist_t* old_distances;
    TwoLevelQueue<vert_t> queue;

    OPERATOR(Vertex& vertex)
    {
        auto child = vertex.id();

        if(d_distances[child] != old_distances[child])
        {
            old_distances[child] = d_distances[child];
            queue.insert(child);
        }
    }
};

struct BFSDynamicDeletion
{
    TwoLevelQueue<vert_t> queue2;

    OPERATOR(Vertex& vertex, Edge& edge) 
    {
        auto dst = edge.dst_id();
        queue2.insert(dst);
    }
};

struct BFSDynamicInsertion
{
    dist_t* d_distances;
    dist_t* old_distances;
    TwoLevelQueue<vert_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge) 
    {
        auto dst = edge.dst_id();
        auto src = vertex.id();

        if(atomicMin(d_distances+dst, d_distances[src]+1) > d_distances[src]+1)
        {
            old_distances[dst] = d_distances[dst];
            queue.insert(dst);
        }
    }
};

struct BFSOperatorCheck
{
    dist_t* d_distances;
    dist_t* old_distances;
    dist_t current_level;
    TwoLevelQueue<vert_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge)
    {
        auto dst = edge.dst_id();
        auto src = vertex.id();

        if (atomicCAS(d_distances + dst, INF, current_level) == INF)
        {
            if(old_distances[dst] != d_distances[dst])
            {
                printf("changed vertex %d old is %d correct is %d father is %d and dd is %d \n",dst,old_distances[dst],d_distances[dst],src,d_distances[src]);
            }
            queue.insert(dst);
        }
    }
};


struct BFSOperatorAtomic 
{                  //deterministic
    dist_t               current_level;
    dist_t*              d_distances;
    dist_t*              old_distances;
    TwoLevelQueue<vid_t> queue;

    OPERATOR(Vertex& vertex, Edge& edge) 
    {
        auto dst = edge.dst_id();
        // if(d_distances[dst]==INF){
        if (atomicCAS(d_distances + dst, INF, current_level) == INF)
        {
            queue.insert(dst);            
            old_distances[dst] = d_distances[dst];
        }
        // }
    }
};

struct PrintCheck
{
    dist_t* d_distances;

    OPERATOR(Vertex& vertex, Edge& edge)
    {
        auto dst = edge.dst_id();
        auto src = vertex.id();

        printf("src %d srcdeg %d dst %d srcdist %d dstdist %d\n",src,vertex.degree(),dst,d_distances[src],d_distances[dst]);
    }
};

//------------------------------------------------------------------------------
/////////////////
// DynBFS //
/////////////////

DynBFS::DynBFS(HornetGraph& hornet, HornetGraph& hornet_in) :
                                    StaticAlgorithm<HornetGraph>(hornet),
                                    queue(hornet, 5),
                                    queue2(hornet, 5),
                                    load_balancing(hornet),
                                    lrb_lb(hornet),
                                    inverted_graph(hornet_in) 
{
    pool.allocate(&d_distances, hornet.nV());
    pool.allocate(&old_distances, hornet.nV());
    reset();
}

DynBFS::~DynBFS() 
{
}

void DynBFS::reset()
{
    current_level = 1;
    queue.clear();

    auto distances = d_distances;
    lb_mechansim = 0;
    forAllnumV(StaticAlgorithm<HornetGraph>::hornet, [=] __device__ (int i){ distances[i] = INF; });

    distances = old_distances;
    forAllnumV(StaticAlgorithm<HornetGraph>::hornet, [=] __device__ (int i){ distances[i] = INF; });
}

void DynBFS::set_parameters(vid_t source,int load_balancing)
{
    bfs_source = source;
    queue.insert(bfs_source);               // insert bfs source in the frontier
    gpu::memsetZero(d_distances + bfs_source);  //reset source distance
    lb_mechansim = load_balancing;
}

void DynBFS::print_check(vert_t* update_src, vert_t* update_dst, int update_size)
{
    queue.clear();
    queue.insert(update_src, update_size);

    std::cout<<"check src queue"<<"\n";

    forAllEdges(StaticAlgorithm<HornetGraph>::hornet, queue, PrintCheck {d_distances}, lrb_lb);

    queue.clear();
    queue.insert(update_dst, update_size);
    std::cout<<"check dst queue"<<"\n";

    forAllEdges(StaticAlgorithm<HornetGraph>::hornet, queue, PrintCheck {d_distances}, lrb_lb);
}

void DynBFS::batch_update_directed(vert_t* update_dst,int update_size)
{
    queue.clear(); //bfs queue clear
    queue2.clear(); //temp queue clear
    queue2.insert(update_dst, update_size); //insert dst vertices in queue

    std::cout<<"VERTICES IN UPDATE QUEUE "<<queue2.size()<<"\n";

    // forAllVertices(inverted_graph, queue2, ResetDistance{d_distances}); //reset distances for each interested vertices in batch
    forAllEdges(inverted_graph, queue2, FatherUpdate {d_distances, old_distances}, lrb_lb); //find lowest father for each modified vertex
    forAllVertices(inverted_graph, queue2, LowestFather {d_distances, old_distances, queue}); //update distance based on lowest father
}

void DynBFS::batch_update_undirected(vert_t* update_src, vert_t* update_dst, int update_size)
{
    //UPDATE DISTANCES BASED ON SRC
    queue.clear(); //bfs queue clear
    queue.insert(update_src,update_size);
    queue.insert(update_src,update_size);
    // queue2.clear(); //temp queue clear
    // queue2.insert(update_src,update_size); //insert update batch in temp queue

    // std::cout<<"VERTICES IN UPDATE QUEUE "<<queue2.size()<<"\n";

    // forAllEdges(hornet, queue2, FatherUpdate {d_distances, old_distances}, lrb_lb); //find lowest father for each modified vertex
    // forAllVertices(hornet, queue2, LowestFather {d_distances, old_distances, queue}); //update distance based on lowest father

    // //UPDATE DISTANCES BASED ON DST 
    // queue2.clear();
    // queue2.insert(update_dst,update_size);

    // std::cout<<"VERTICES IN UPDATE QUEUE "<<queue2.size()<<"\n";

    // forAllEdges(hornet, queue2, FatherUpdate {d_distances, old_distances}, lrb_lb);
    // forAllVertices(hornet, queue2, LowestFather {d_distances, old_distances, queue});
}

void DynBFS::run()
{
    printf("bfs_source = %d\n",bfs_source);

    while (queue.size() > 0) {
    
        if(lb_mechansim==0){
            forAllEdges(
                StaticAlgorithm<HornetGraph>::hornet, queue,
                        BFSOperatorAtomic { current_level, d_distances, old_distances, queue },
                        lrb_lb);
        }else{
            forAllEdges(
                StaticAlgorithm<HornetGraph>::hornet, queue,
                        BFSOperatorAtomic { current_level, d_distances, old_distances, queue },
                        load_balancing);
        }
        queue.swap();
        current_level++;
    }
    // std::cout << "Number of levels is : " << current_level << std::endl;
}

void DynBFS::run_dynamic()
{
    int count = 0;
    int totalVisited = 0;

    queue.swap();

    // if(!batch_type)
    // {
    while(queue.size() > 0)
    {
        std::cout<<"VERTICES IN QUEUE "<<queue.size()<<"\n";
        totalVisited += queue.size();

        //execute BFS
        forAllEdges(hornet, queue, BFSDynamicInsertion {d_distances, old_distances, queue}, lrb_lb);
        queue.swap();
    
        ++count;
    }
    // }
    // else
    // {
    //     while(queue.size() > 0)
    //     {
    //         std::cout<<"VERTICES IN QUEUE "<<queue.size()<<"\n";
    //         totalVisited += queue.size();

    //         //execute BFS
    //         forAllEdges(hornet, queue, BFSDynamicDeletion {queue2}, load_balancing);

    //         queue2.swap();
    //         forAllVertices(hornet, queue2, ResetDistance{d_distances}); //reset distances for each interested vertices in batch
    //         forAllEdges(hornet, queue2, FatherUpdate {d_distances, old_distances}, load_balancing); //find lowest father for each modified vertex
    //         forAllVertices(hornet, queue2, LowestFather {d_distances, old_distances, queue}); //update distance based on lowest father
    //         queue.swap();

    //         ++count;
    //     }
    // }

    std::cout<<"BFS EXECUTED "<<count<<" TIMES AND VISITED "<<totalVisited<<" NODES\n";
}

void DynBFS::run_check(vid_t source)
{
    current_level = 1;
    auto distances = d_distances;
    forAllnumV(StaticAlgorithm<HornetGraph>::hornet, [=] __device__ (int i){ distances[i] = INF; });

    bfs_source = source;
    queue.clear();
    queue.insert(bfs_source);
    gpu::memsetZero(d_distances + bfs_source);  //reset source distance

    while(queue.size() > 0)
    {
        forAllEdges(hornet, queue, BFSOperatorCheck {d_distances, old_distances, current_level, queue }, lrb_lb);
        queue.swap();
        current_level++;
    }
}


void DynBFS::release()
{
    d_distances = nullptr;
}

bool DynBFS::validate()
{
    return true;
}

} // namespace hornets_nest
