/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off
// #include "Dynamic/BFS/DynBFS.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off
#include <Hornet.hpp>
#include "StandardAPI.hpp"
#include "Util/BatchFunctions.hpp"
#include "Util/RandomGraphData.cuh"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <Core/Static/Static.cuh>
#include <ctime>

template <typename HornetGraph, typename BFS>
int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;
    using HornetGPU = hornet::gpu::Hornet<vert_t>;
    using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::EMPTY, hornet::DeviceType::HOST>;
    using BatchUpdate = hornet::gpu::BatchUpdate<vert_t>;

    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1000000000000000000);


    // graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED );
    // graph::GraphStd<vid_t, eoff_t> graph(DIRECTED);
    // graph::GraphStd<vid_t, eoff_t> graph(ENABLE_INGOING);
    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);
    // ParsingProp pp;
    // graph.read(argv[1],pp);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    Timer<DEVICE> TM;
    HornetGraph hornet_graph(hornet_init);

    BFS bfs_top_down(hornet_graph);

    vid_t root = graph.max_out_degree_id();
    // if (argc==3)
    //     root = atoi(argv[2]);
    int numberRoots = 1;
    if (argc>=3)
      numberRoots = atoi(argv[2]);

    int alg = 0;
    if (argc>=4)
      alg = atoi(argv[3]);

    std::cout << "My root is " << root << std::endl;

    std::cout<<"prima size "<<hornet_graph.nE()<<"\n";


    int batch_size = 10000;

    vert_t* batch_src = new vert_t[batch_size]();
    vert_t* batch_dst = new vert_t[batch_size]();

    generateBatch(graph, batch_size, batch_src, batch_dst, BatchGenType::INSERT, batch_gen_property::UNIQUE);

    UpdatePtr ptr(batch_size, batch_src, batch_dst);
    BatchUpdate batch_update_src_to_dst(ptr);

    UpdatePtr ptr2(batch_size, batch_dst, batch_src);
    BatchUpdate batch_update_dst_to_src(ptr2);

    hornet_graph.insert(batch_update_src_to_dst);
    hornet_graph.insert(batch_update_dst_to_src);

    batch_update_src_to_dst.print();

    std::cout<<"dopo size "<<hornet_graph.nE()<<"\n";

    hipProfilerStart();
    for(int i=0; i<numberRoots; i++){
        bfs_top_down.reset();
        bfs_top_down.set_parameters((root+i)%graph.nV(),alg);
    TM.start();
        // bfs_top_down.run();
        bfs_top_down.print_check(batch_src, batch_dst, batch_size);
    TM.stop();
        std::cout << "Number of levels is : " << bfs_top_down.getLevels() << std::endl;
    }

    hipProfilerStop();
    TM.print("TopDown2");

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.

    ret = exec<hornets_nest::HornetDynamicGraph, hornets_nest::BfsTopDown2Dynamic>(argc, argv);
    // ret = exec<hornets_nest::HornetStaticGraph,  hornets_nest::BfsTopDown2Static >(argc, argv);

    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();

    return ret;
}

