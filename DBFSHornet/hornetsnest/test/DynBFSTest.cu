/**
* @brief Breadth-first Search Top-Down test program
* @file
*/
#include "Dynamic/BFS/DynBFS.cuh"
#include "StandardAPI.hpp"
#include "Static/BreadthFirstSearch/TopDown2.cuh"
#include "Util/BatchFunctions.hpp"
#include "Util/RandomGraphData.cuh"
#include <Core/Static/Static.cuh>
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <Graph/GraphStd.hpp>
#include <Hornet.hpp>
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <StandardAPI.hpp>
#include <Util/CommandLineParam.hpp>
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <ctime>
#include <hip/hip_runtime_api.h> //--profile-from-start off
#include <random>                       //std::mt19937_64

template <typename HornetGraph, typename BFS>
int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;
    using HornetGPU = hornet::gpu::Hornet<vert_t>;
    using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::EMPTY, hornet::DeviceType::HOST>;
    // using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::EMPTY, hornet::DeviceType::DEVICE>;
    using BatchUpdate = hornet::gpu::BatchUpdate<vert_t>;

    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1000000000000000000);

    graph::GraphStd<vid_t, eoff_t> graph(ENABLE_INGOING);
    CommandLineParam cmd(graph, argc, argv,false);

    HornetInit hornet_init_out(graph.nV(), graph.nE(), graph.csr_out_offsets(), graph.csr_out_edges());
    HornetInit hornet_init_in(graph.nV(), graph.nE(), graph.csr_in_offsets(), graph.csr_in_edges());

    Timer<DEVICE> TM;
    HornetGraph hornet_graph_out(hornet_init_out);
    HornetGraph hornet_graph_in(hornet_init_in);

    DynBFS dynamic_bfs(hornet_graph_out, hornet_graph_in);

    vid_t root = graph.max_out_degree_id();
    int alg = 0;
    int batch_size = 0;

    if (argc >= 3)
        root = atoi(argv[2]);
    if(argc >= 4)
        batch_size = atoi(argv[3]);

    dynamic_bfs.reset();
    dynamic_bfs.set_parameters(root,alg);

    std::cout << "My root is " << root << std::endl;

    TM.start();
    dynamic_bfs.run();
    TM.stop();
    TM.print("Static BFS");

    std::cout << "Number of levels is : " << dynamic_bfs.getLevels() << std::endl;

    std::cout<<"prima size "<<hornet_graph_out.nE()<<"\n";

    vert_t* batch_src = new vert_t[batch_size]();
    vert_t* batch_dst = new vert_t[batch_size]();

    generateBatch(graph, batch_size, batch_src, batch_dst, BatchGenType::INSERT, batch_gen_property::UNIQUE);

    UpdatePtr ptr(batch_size, batch_src, batch_dst);
    BatchUpdate batch_update_src_to_dst(ptr);

    UpdatePtr ptr2(batch_size, batch_dst, batch_src);
    BatchUpdate batch_update_dst_to_src(ptr2);

    if(graph.is_directed())
    {
        std::cout<<"entro directed "<<"\n";
        hornet_graph_in.insert(batch_update_dst_to_src);
        hornet_graph_out.insert(batch_update_src_to_dst);

    }
    else
    {
        hornet_graph_out.insert(batch_update_src_to_dst);
        hornet_graph_out.insert(batch_update_dst_to_src);
    }

    // batch_update_src_to_dst.print();
    // batch_update_dst_to_src.print();

    std::cout<<"dopo size "<<hornet_graph_out.nE()<<"\n";

    // hipProfilerStart();

    TM.start();
    // dynamic_bfs.run();
    // dynamic_bfs.print_check(batch_src, batch_dst, batch_size);

    if(graph.is_directed())
        dynamic_bfs.batch_update_directed(batch_dst,batch_size);
    else
        dynamic_bfs.batch_update_undirected(batch_src,batch_dst,batch_size);

    dynamic_bfs.run_dynamic();

    TM.stop();
    TM.print("Dynamic BFS");


    TM.start();
    dynamic_bfs.run_check(root);
    TM.stop();
    TM.print("BFS check");

    dynamic_bfs.reset();
    dynamic_bfs.set_parameters(root,alg);

    std::cout << "My root is " << root << std::endl;

    TM.start();
    dynamic_bfs.run();
    TM.stop();
    TM.print("Static BFS recheck");


    std::cout << "Number of levels is : " << dynamic_bfs.getLevels() << std::endl;

    // hipProfilerStop();
    // TM.print("TopDown2");

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.

    ret = exec<hornets_nest::HornetDynamicGraph, hornets_nest::BfsTopDown2Dynamic>(argc, argv);
    // ret = exec<hornets_nest::HornetStaticGraph,  hornets_nest::BfsTopDown2Static >(argc, argv);

    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();

    return ret;
}

